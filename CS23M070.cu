#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include<thrust/copy.h>

#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>
#define MAX_DIST 1e18
using namespace std;

//*******************************************

// Write down the kernels here
__device__ long long min_val;

__device__ bool comp(long long x1, long long y1, long long x2, long long y2, long long x3, long long y3){
    bool flgx=false, flgy=false;
    if((x1>=x2 && x1>=x3) || (x1<=x2 && x1<=x3)) flgx=true;
    if((y1>=y2 && y1>=y3) || (y1<=y2 && y1<=y3)) flgy=true;
    return (flgx && flgy);
}
                                                                          
__global__ void reduce_health(long long *SCORE, int *HEALTH, long long *HIT, int *sz, int T){
    long long attacker = threadIdx.x;
    long long victim = HIT[attacker];
    if(victim!=T){
        SCORE[attacker]+=1;
        long long old_health = atomicAdd(&HEALTH[victim],-1);
        if(old_health==1) atomicAdd(sz,-1);
    }
}

__global__ void copy_health_and_score(int *HEALTH, long long *SNAP_HEALTH, long long *SCORE, long long *SNAP_SCORE, int T){
    long long id = threadIdx.x;
    SNAP_HEALTH[id] = HEALTH[id];
    SNAP_SCORE[id] = SCORE[id];
}
__global__ void play_fast(int *HEALTH, long long *SNAP_HEALTH, long long *SCORE, long long *SNAP_SCORE, int T){
    long long id = threadIdx.x;
    if(id==0) min_val = MAX_DIST;
    __syncthreads();
    if(HEALTH[id]>0){
        long long current_health = HEALTH[id];
        long long health_change = SNAP_HEALTH[id] - current_health;
        long long rounds_before_death = ceil(current_health/(float)health_change) - 1;
        atomicMin(&min_val, rounds_before_death);
    }
    __syncthreads();
    if(HEALTH[id]>0){
        long long current_health = HEALTH[id];
        long long health_change = SNAP_HEALTH[id] - current_health;
        long long current_score = SCORE[id];
        long long score_change = current_score - SNAP_SCORE[id];
        SCORE[id]+=(score_change * min_val);
        HEALTH[id]-=(health_change * min_val);
    }
    //if(id==0) printf("Fast Forward --> %d\n",min_val);
}

__global__ void play_game(long long *XCOORD, long long *YCOORD, long long *SCORE, int *HEALTH, long long *HIT, long long *DIST, int gap, int T){
    long long attacker = blockIdx.x, new_victim = threadIdx.x;
    long long org_victim=(attacker+gap)%T;
    long long xa = XCOORD[attacker], xov = XCOORD[org_victim], xnv = XCOORD[new_victim];
    long long ya = YCOORD[attacker], yov = YCOORD[org_victim], ynv = YCOORD[new_victim];
    
    if(new_victim==0){
        DIST[attacker] = MAX_DIST;
        HIT[attacker] = T;
    }
    __syncthreads();
    long long non_collinear = xa*(yov-ynv) + xov*(ynv-ya) + xnv*(ya-yov);
    long long comp_dist = MAX_DIST;
    if(new_victim!=attacker && HEALTH[attacker]>0 && HEALTH[new_victim]>0 && non_collinear==0 && comp(xa,ya,xov,yov,xnv,ynv)){    
        long long x=(xa-xnv),y=(ya-ynv);
        comp_dist = x*x+y*y;
        atomicMin(&DIST[attacker],comp_dist);
    }
    __syncthreads();
    if(comp_dist!=MAX_DIST && comp_dist==DIST[attacker]) HIT[attacker]=new_victim;  
}

//***********************************************


int main(int argc,char **argv)
{
    // Variable declarations
    int M,N,T,H,*xcoord,*ycoord,*score;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &M );
    fscanf( inputfilepointer, "%d", &N );
    fscanf( inputfilepointer, "%d", &T ); // T is number of Tanks
    fscanf( inputfilepointer, "%d", &H ); // H is the starting Health point of each Tank
	
    // Allocate memory on CPU
    xcoord=(int*)malloc(T * sizeof (int));  // X coordinate of each tank
    ycoord=(int*)malloc(T * sizeof (int));  // Y coordinate of each tank
    score=(int*)malloc(T * sizeof (int));  // Score of each tank (ensure that at the end you have copied back the score calculations on the GPU back to this allocation)

    // Get the Input of Tank coordinates
    for(int i=0;i<T;i++)
    {
      fscanf( inputfilepointer, "%d", &xcoord[i] );
      fscanf( inputfilepointer, "%d", &ycoord[i] );
    }
		

    auto start = chrono::high_resolution_clock::now();

    //*********************************
    // Your Code begins here (Do not change anything in main() above this comment)
    //********************************

    thrust::device_vector<long long> XCOORD(T), YCOORD(T), SCORE(T,0), HIT(T,T), SNAP_SCORE(T,0), SNAP_HEALTH(T,H),DIST(T);
    thrust::device_vector<int> HEALTH(T,H);
    thrust::copy(xcoord, xcoord+T, XCOORD.begin());
    thrust::copy(ycoord, ycoord+T, YCOORD.begin());

    int *sz;
    hipHostAlloc(&sz, sizeof(int), 0);
    *sz=T;

    for(int gap=0, prev_sz=T;*sz>1;gap=(gap+1)%T){
        if(gap==0){
            copy_health_and_score<<<1,T>>>(thrust::raw_pointer_cast(HEALTH.data()),thrust::raw_pointer_cast(SNAP_HEALTH.data()),thrust::raw_pointer_cast(SCORE.data()),thrust::raw_pointer_cast(SNAP_SCORE.data()),T);
            prev_sz = *sz;
            continue;
        }
        play_game<<<T,T>>>(thrust::raw_pointer_cast(XCOORD.data()), thrust::raw_pointer_cast(YCOORD.data()), thrust::raw_pointer_cast(SCORE.data()), thrust::raw_pointer_cast(HEALTH.data()), thrust::raw_pointer_cast(HIT.data()), thrust::raw_pointer_cast(DIST.data()), gap, T);
        reduce_health<<<1,T>>>(thrust::raw_pointer_cast(SCORE.data()),thrust::raw_pointer_cast(HEALTH.data()),thrust::raw_pointer_cast(HIT.data()),sz,T);
        hipDeviceSynchronize();
        if(gap==T-1 && prev_sz==*sz) play_fast<<<1,T>>>(thrust::raw_pointer_cast(HEALTH.data()), thrust::raw_pointer_cast(SNAP_HEALTH.data()),thrust::raw_pointer_cast(SCORE.data()), thrust::raw_pointer_cast(SNAP_SCORE.data()), T);
    }
    thrust::copy(SCORE.begin(), SCORE.end(), score);

    //*********************************
    // Your Code ends here (Do not change anything in main() below this comment)
    //********************************

    auto end  = chrono::high_resolution_clock::now();

    chrono::duration<double, std::micro> timeTaken = end-start;

    printf("Execution time : %f\n", timeTaken.count());

    // Output
    char *outputfilename = argv[2];
    char *exectimefilename = argv[3]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    for(int i=0;i<T;i++)
    {
        fprintf( outputfilepointer, "%d\n", score[i]);
    }
    fclose(inputfilepointer);
    fclose(outputfilepointer);

    outputfilepointer = fopen(exectimefilename,"w");
    fprintf(outputfilepointer,"%f", timeTaken.count());
    fclose(outputfilepointer);

    free(xcoord);
    free(ycoord);
    free(score);
    hipDeviceSynchronize();
    return 0;
}